#include "hip/hip_runtime.h"
// kernel.cu
#include "kernel.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <chrono>

using namespace std;
using namespace std::chrono;

__global__ void vec_add_kernel(float *a, float *b, float *c, int n) {
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    if (i < n) c[i] = a[i] + b[i];
}

int cuda_vec_add(float *h_a, float *h_b, float *h_c, int n) {
    float *d_a, *d_b, *d_c;

    hipMalloc(&d_a, n*sizeof(float));
    hipMalloc(&d_b, n*sizeof(float));
    hipMalloc(&d_c, n*sizeof(float));

    hipMemcpy(d_a, h_a, n*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, n*sizeof(float), hipMemcpyHostToDevice);

    auto beg_cuda = steady_clock::now();
    vec_add_kernel<<<1,1>>>(d_a, d_b, d_c, n);
    auto end_cuda = steady_clock::now();

    cout << "Elapsed Time [CUDA]: " << std::chrono::duration_cast<std::chrono::microseconds>(end_cuda-beg_cuda).count()
        << " [us]" << endl;

    hipMemcpy(h_c, d_c, n*sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_a); hipFree(d_b); hipFree(d_c);

    return 0;
}