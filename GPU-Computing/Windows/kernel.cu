#include "hip/hip_runtime.h"
#include "kernel.h"
#include <stdio.h>

#define TPB 1024  // Threads per Block

__device__
float distance(float x1, float x2)
{
    return sqrt((x2-x1)*(x2-x1));
}

__global__
void distanceKernel(float *d_out, float *d_in, float ref)
{
    const int i = blockIdx.x*blockDim.x + threadIdx.x;
    const float x = d_in[i];
    d_out[i] = distance(x, ref);
}

void distanceArray(float *out, float *in, float ref, unsigned long len)
{
    float *d_in = 0;
    float *d_out = 0;

    hipMalloc(&d_in, len*sizeof(float));
    hipMalloc(&d_out, len*sizeof(float));

    hipMemcpy(d_in, in, len*sizeof(float), hipMemcpyHostToDevice);
    distanceKernel<<<len/TPB, TPB>>>(d_out, d_in, ref);

    hipMemcpy(out, d_out, len*sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_in);
    hipFree(d_out);
    
}